#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

#include <opencv2\opencv.hpp>
#include <opencv2/core/core.hpp> 
#include<opencv2/highgui/highgui.hpp>  

#include <iostream>
using namespace std;
using namespace cv;

//Sobel���ӱ�Ե���˺���
__global__ void sobelInCuda(unsigned char *dataIn, unsigned char *dataOut, int imgHeight, int imgWidth)
{
    int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    int index = yIndex * imgWidth + xIndex;
    int Gx = 0;
    int Gy = 0;

    if (xIndex > 0 && xIndex < imgWidth - 1 && yIndex > 0 && yIndex < imgHeight - 1)
    {
        Gx = dataIn[(yIndex - 1) * imgWidth + xIndex + 1] + 2 * dataIn[yIndex * imgWidth + xIndex + 1] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[yIndex * imgWidth + xIndex - 1] + dataIn[(yIndex + 1) * imgWidth + xIndex - 1]);
        Gy = dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex - 1) * imgWidth + xIndex] + dataIn[(yIndex - 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex + 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex + 1) * imgWidth + xIndex] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]);
        dataOut[index] = (abs(Gx) + abs(Gy)) / 2;
    }
}

//Sobel���ӱ�Ե���CPU����
void sobel(Mat srcImg, Mat dstImg, int imgHeight, int imgWidth)
{
    int Gx = 0;
    int Gy = 0;
    for (int i = 1; i < imgHeight - 1; i++)
    {
        uchar *dataUp = srcImg.ptr<uchar>(i - 1);
        uchar *data = srcImg.ptr<uchar>(i);
        uchar *dataDown = srcImg.ptr<uchar>(i + 1);
        uchar *out = dstImg.ptr<uchar>(i);
        for (int j = 1; j < imgWidth - 1; j++)
        {
            Gx = (dataUp[j + 1] + 2 * data[j + 1] + dataDown[j + 1]) - (dataUp[j - 1] + 2 * data[j - 1] + dataDown[j - 1]);
            Gy = (dataUp[j - 1] + 2 * dataUp[j] + dataUp[j + 1]) - (dataDown[j - 1] + 2 * dataDown[j] + dataDown[j + 1]);
            out[j] = (abs(Gx) + abs(Gy)) / 2;
        }
    }
}

int main()
{
   Mat grayImg = imread("1.jpg", 0);
    
   if (!grayImg.data) {
       std::cout << "Image not loaded";
       return -1;
   }

   namedWindow("windows", CV_WINDOW_AUTOSIZE);
   imshow("windows", grayImg);


    imwrite("Gray_Image.jpg", grayImg);


    int imgHeight = grayImg.rows;
    int imgWidth = grayImg.cols;

    Mat gaussImg;
    //��˹�˲�
    GaussianBlur(grayImg, gaussImg, Size(3, 3), 0, 0, BORDER_DEFAULT);

    //Sobel����CPUʵ��
    Mat dst(imgHeight, imgWidth, CV_8UC1, Scalar(0));
    sobel(gaussImg, dst, imgHeight, imgWidth);

    //CUDAʵ�ֺ�Ĵ��ص�ͼ��
    Mat dstImg(imgHeight, imgWidth, CV_8UC1, Scalar(0));

    //����GPU�ڴ�
    unsigned char *d_in;
    unsigned char *d_out;

    hipMalloc((void**)&d_in, imgHeight * imgWidth * sizeof(unsigned char));
    hipMalloc((void**)&d_out, imgHeight * imgWidth * sizeof(unsigned char));

    //����˹�˲����ͼ���CPU����GPU
    hipMemcpy(d_in, gaussImg.data, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, (imgHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);

    //���ú˺���
    sobelInCuda << < blocksPerGrid, threadsPerBlock >> > (d_in, d_out, imgHeight, imgWidth);

    //��ͼ�񴫻�GPU
    hipMemcpy(dstImg.data, d_out, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyDeviceToHost);

    imwrite("dst_Image.jpg", dst);

    imwrite("dstImg_Image.jpg", dstImg);

    //�ͷ�GPU�ڴ�
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
